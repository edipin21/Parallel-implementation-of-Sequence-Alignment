#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdlib.h>
#include "omp.h"
#define NUM_OF_WEIGHTS 4
#define SIZE_OF_CONS 9
#define SIZE_OF_S_CONS 11
__constant__ char conservativeGrope[SIZE_OF_CONS][5] = {"NDEQ", "NEQK", "STA", "MILV", "QHRK", "NHQK", "FYW", "HY", "MILF"};
__constant__ char semiConservativeGroup[SIZE_OF_S_CONS][7] = {"SAG", "ATV", "CSA", "SGND", "STPA", "STNK", "NEQHRK", "NDEQHK", "SNDEQK", "HFY", "FVLIM"};

__device__ int onGroup(char c1, char c2, int grpIND)
{
    int j, count;
    char currLetter;

    if (grpIND == 0)
    {

        for (int i = 0; i < SIZE_OF_CONS; i++)
        {
            j = 0;
            count = 0;
            currLetter = conservativeGrope[i][j];

            while (currLetter != '\0')
            {
                if (c1 == currLetter || c2 == currLetter)
                    count++;
                if (count == 2)
                    return 1;
                j++;
                currLetter = conservativeGrope[i][j];
            }
        }
    }
    else
    {
        for (int i = 0; i < SIZE_OF_S_CONS; i++)
        {
            j = 0;
            count = 0;
            currLetter = semiConservativeGroup[i][j];

            while (currLetter != '\0')
            {
                if (c1 == currLetter || c2 == currLetter)
                    count++;
                if (count == 2)
                    return 1;
                j++;
                currLetter = semiConservativeGroup[i][j];
            }
        }
    }

    return 0;
}
__global__ void getArrBestScorePermut(const char *seq2, int seq2Len, const char *Seq1, int *IndexsToMakeMut, int numOfmut, int numOfAlignment, double *weights, double *BestScorePerMut, int *bestAlignment)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numOfmut)
    {
        BestScorePerMut[i] = 0;

        for (int j = 0; j < numOfAlignment; j++)
        {
            int corectseq1Aligment = 0;
            double currScore = 0;
            for (int k = 0; k < seq2Len; k++)
            {
                if (k == IndexsToMakeMut[i + i] || k == IndexsToMakeMut[i + i + 1])

                    corectseq1Aligment++;

                else
                {
                    if (seq2[k] == Seq1[k + j - corectseq1Aligment])
                        currScore += weights[0];

                    else if (onGroup(seq2[k], Seq1[k + j - corectseq1Aligment], 0) == 1)
                        currScore -= weights[1];

                    else if (onGroup(seq2[k], Seq1[k + j - corectseq1Aligment], 1) == 1)
                        currScore -= weights[2];

                    else
                        currScore -= weights[3];
                }
            }
            if (currScore > BestScorePerMut[i])
            {

                BestScorePerMut[i] = currScore;
                bestAlignment[i] = j;
            }
        }
    }
}

int *getSeqBestMsAndOffsetOnGpu(char *seq2, char *seq1, int seq1Len, int seq2Len, int *allIndexsToMakeMut, double numOfMut, int numOfAlignment, double *weights, int *bestOffset)
{

    hipError_t err = hipSuccess;

    char *d_seq1, *d_seq2;
    int *d_allIndexsToMakeMut, *d_bestAlignmentPerMut;
    double *d_weights, *d_BestScorePerMut;

    // Allocate the device input seq2
    err = hipMalloc((void **)&d_seq2, seq2Len * sizeof(char));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device seq2 (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    // Allocate the device input mutant indexs
    err = hipMalloc((void **)&d_allIndexsToMakeMut, (numOfMut * 2) * sizeof(int));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device mutant indexs (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    // Allocate the device input seq1
    err = hipMalloc((void **)&d_seq1, seq1Len * sizeof(char));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device seq1 (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    // Allocate the device input best alignment array
    err = hipMalloc((void **)&d_bestAlignmentPerMut, numOfMut * sizeof(int));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device best alignment array (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    // Allocate the device input weights
    err = hipMalloc((void **)&d_weights, NUM_OF_WEIGHTS * sizeof(double));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device weights array (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    // Allocate the device input best score per mutant
    err = hipMalloc((void **)&d_BestScorePerMut, numOfMut * sizeof(double));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device best score per mutant array (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    // Copy the host input seq2  in host memory to the device input vectors in device memory
    err = hipMemcpy(d_seq2, seq2, seq2Len, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy seq2 from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    // Copy the host input indexs array  in host memory to the device input vectors in device memory
    err = hipMemcpy(d_allIndexsToMakeMut, allIndexsToMakeMut, (numOfMut * 2) * sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy indexs array from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    // Copy the host input seq1 in host memory to the device input vectors in device memory
    err = hipMemcpy(d_seq1, seq1, seq1Len, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy seq1 from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    // Copy the host input weights in host memory to the device input vectors in device memory
    err = hipMemcpy(d_weights, weights, NUM_OF_WEIGHTS * sizeof(double), hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy weights from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    int threadsPerBlock = 1024;
    int blocksPerGrid = (numOfMut + threadsPerBlock - 1) / threadsPerBlock;
    double *h_BestScorePerMut = (double *)malloc(sizeof(double) * numOfMut);
    int *h_bestOffset = (int *)malloc(sizeof(int) * numOfMut);
    getArrBestScorePermut<<<blocksPerGrid, threadsPerBlock>>>(d_seq2, seq2Len, d_seq1, d_allIndexsToMakeMut, numOfMut, numOfAlignment, d_weights, d_BestScorePerMut, d_bestAlignmentPerMut);

    // Copy the device input best score per mutant array in device memory to the host input vectors in host memory
    err = hipMemcpy(h_BestScorePerMut, d_BestScorePerMut, numOfMut * sizeof(double), hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy BestScorePerMut array from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    // Copy the device input best offset per mutant array in device memory to the host input vectors in host memory
    err = hipMemcpy(h_bestOffset, d_bestAlignmentPerMut, numOfMut * sizeof(int), hipMemcpyDeviceToHost);

    double bs = 0;
    int indexOfBestMsAndOffset;

#pragma omp parallel for
    for (int i = 0; i < numOfMut; i++)
    {
        if (h_BestScorePerMut[i] > bs)
        {
            bs = h_BestScorePerMut[i];
            indexOfBestMsAndOffset = i;
        }
    }
    *bestOffset = h_bestOffset[indexOfBestMsAndOffset];
    int *MS = (int *)malloc(sizeof(int) * 2);
    MS[0] = allIndexsToMakeMut[indexOfBestMsAndOffset + indexOfBestMsAndOffset] + 1;
    MS[1] = allIndexsToMakeMut[indexOfBestMsAndOffset + indexOfBestMsAndOffset + 1] + 1;
    
    // Free device global memory
    err = hipFree(d_allIndexsToMakeMut);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device allIndexsToMakeMut  (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_bestAlignmentPerMut);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device bestAlignmentPerMut  (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipFree(d_BestScorePerMut);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device BestScorePerMut  (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipFree(d_seq1);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device seq1  (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_seq2);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device seq2  (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipFree(d_weights);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device weights  (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free host memory
    free(h_bestOffset);
    free(h_BestScorePerMut);

    return MS;
}